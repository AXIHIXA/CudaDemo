#include "hip/hip_runtime.h"
#include <algorithm>
#include <chrono>
#include <cmath>
#include <cstdlib>
#include <iostream>
#include <numeric>
#include <random>
#include <vector>

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/host_vector.h>

#include "utils/cuda_utils.h"


/// GEMM kernel for fp32 with cuBLAS, calculating y == alpha * (A @ x) + beta.
/// Used for ground truth calculation.
/// cuBLAS matrices are in COLUMN-MAJOR, A should be transposed.
/// \param[in] A      shape=(dm, dn)
/// \param[in/out] B  shape=(dn, 1)
void gemvCublas(const float * __restrict__ a,
                const float * __restrict__ x,
                float * __restrict__ y,
                int dm,
                int dn,
                float alpha,
                float beta,
                hipblasHandle_t handle)
{
    CUBLAS_CHECK(
        hipblasSgemv(
            handle, HIPBLAS_OP_T,
            dm, dn,
            &alpha,
            a, dn,
            x, 1,
            &beta,
            y, 1
        )
    );
}


template <typename acc_t, int kWarpThreads = 32>
__device__ acc_t warpReduce(acc_t val)
{
    static_assert(kWarpThreads == 32);

    #pragma unroll
    for (int step = (kWarpThreads >> 1); 0 < step; step >>= 1)
    {
        val += __shfl_xor_sync(0xffffffff, val, step, kWarpThreads);
    }

    return val;
}


template <int kBlockDimX, int kBlockDimY, int kWarpThreads = 32, typename acc_t>
__device__ acc_t blockReduce(acc_t val)
{
    static_assert(kBlockDimX % kWarpThreads == 0 && kBlockDimY == 1 && kWarpThreads == 32);
    constexpr int kWarps = kBlockDimX / kWarpThreads;
    val = warpReduce(val);

    __shared__ acc_t warpAggregate[kWarps];

    const int laneIdx = threadIdx.x % kWarpThreads;
    const int warpIdx = threadIdx.x / kWarpThreads;

    if (warpIdx < kWarps && laneIdx == 0)
    {
        warpAggregate[warpIdx] = val;
    }

    __syncthreads();

    val = 0;

    #pragma unroll
    for (int warp = 0; warp < kWarps; ++warp)
    {
        val += warpAggregate[warp];
    }

    return val;
}


/// Each 1D block computes one element in output.
template <
        int kBlockDimX, int kBlockDimY,
        int kBlockSpanX, int kBlockSpanY,
        int kPadding = 4,
        typename in_t, typename acc_t>
__global__ void gemvNaive(
        const in_t * __restrict__ a,
        const in_t * __restrict__ x,
        acc_t * __restrict__ y,
        int dm,
        int dn,
        acc_t alpha,
        acc_t beta)
{
    static_assert(std::is_same_v<in_t, float>);
    static_assert(kBlockSpanX % kBlockDimX == 0);
    static_assert(kBlockDimY == 1 && kBlockSpanY == 1);
    constexpr int kThreadSpanX = kBlockSpanX / kBlockDimX;
    constexpr int kPackSize = 1;

    acc_t threadData;

    // Grid Translation.
    for (int gy = blockIdx.y * kBlockSpanY; gy < dm; gy += gridDim.x * kBlockSpanY)
    {
        threadData = 0;

        #pragma unroll
        for (int packIdx = 0; packIdx < kThreadSpanX; ++packIdx)
        {
            // p0 p0 p0 ... p0   p1 p1 p1 ... p1   ...
            const int gx = packIdx * kPackSize * kBlockDimX + threadIdx.x * kPackSize;

            if (gx < dn)
            {
                threadData += a[gy * dn + gx] * x[gx];
            }
        }


    }


}


template <typename T>
struct Equal
{
    __host__ __device__
    inline bool operator()(const T & a, const T & b) = delete;
};


template <>
struct Equal<float>
{
    __host__ __device__
    inline bool operator()(float a, float b)
    {
        return abs(a - b) < kAbsTol + kRelTol * abs(b);
    }

    static constexpr float kAbsTol = 1e-3f;
    static constexpr float kRelTol = 2e-4f;
};


template <bool kDebugOutput = true, typename acc_t>
void checkResult(const thrust::device_vector<acc_t> & result,
                 const thrust::device_vector<acc_t> & golden,
                 int dm)
{
    if constexpr (kDebugOutput)
    {
        thrust::host_vector<acc_t> a = result;
        thrust::host_vector<acc_t> b = golden;

        printf("Result:\n");
        for (int i = 0; i < dm; ++i)
        {
            printf("%f ", a[i]);
            printf("\n");
        }
        printf("\n\n");

        printf("Ground truth:\n");
        for (int i = 0; i < dm; ++i)
        {
            printf("%f ", b[i]);
            printf("\n");
        }
        printf("\n\n");
    }

    bool resultIsCorrect = thrust::equal(thrust::device, result.cbegin(), result.cend(), golden.cbegin(), Equal<acc_t>());
    std::printf("Result is %s\n\n", resultIsCorrect ? "correct." : "WRONG!!!");
}


int main(int argc, char * argv[])
{
    /// Switches for debugging output correctness.
    /// \param kDup        Set to 1 to debug output (kernel only launched once) and results will be checked.
    ///                    Set to values greater than 1 to profile.
    ///                    In the latter case, results will NOT be checked because it's in-place GEMM.
    ///                    We do not dispatch by build type because we have -G flag for Debug builds
    ///                    (that's for debugging runtime errors).
    /// \param kRandInput  Whether we random input matrices.
    ///                    Enable when checking correctness or profiling.
    ///                    Disable when debugging output.
    constexpr int kDup = 1;
    constexpr bool kRandInput = false;

    constexpr bool kTestGemvNaive = true;
    constexpr bool kTestGemvSmem = true;

    // Problem setting.
    // Tested on NVIDIA Geforce RTX 2080 Ti (kDup=100, kRandInput=true),
    // gemmSmemPad reaches 80% hipblasSgemm performance on m=n=k=2048,
    //                     90% hipblasSgemm performance on m=n=k=4096.
    // It shows that HIPBLAS_DEFAULT_MATH defaults to HIPBLAS_TF32_TENSOR_OP_MATH.
    int problemSize = 4;
    int m = problemSize;
    int n = problemSize;
    float alpha = 1.0f;
    float beta = 0.0f;
    thrust::host_vector<float> h_a(m * n, 1.0f);
    thrust::host_vector<float> h_x(m, 1.0f);
    thrust::host_vector<float> h_y(m, 0.0f);
    std::iota(h_a.begin(), h_a.end(), 1.0f);
    std::iota(h_x.begin(), h_x.end(), 1.0f);

    if constexpr (kRandInput)
    {
        unsigned seed = std::random_device()();
        std::default_random_engine e(seed);
        // std::normal_distribution<float> d(0.0f, 1.0f);
        std::uniform_int_distribution d(1, 20);
        auto g = [&e, &d]() -> float { return d(e); };
        std::generate(h_a.begin(), h_a.end(), g);
        std::generate(h_x.begin(), h_x.end(), g);
        std::generate(h_y.begin(), h_y.end(), g);
    }

    thrust::device_vector<float> golden_y(n);
    thrust::device_vector<float> d_a = h_a;
    thrust::device_vector<float> d_x = h_x;
    thrust::device_vector<float> d_y = h_y;

    // CUDA resources that require manual destruction.
    float ms;
    hipEvent_t ss, ee;
    CUDA_CHECK(hipEventCreate(&ss));
    CUDA_CHECK(hipEventCreate(&ee));

    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    // Testing says that these two modes are the same.
    CUBLAS_CHECK(hipblasSetMathMode(handle, HIPBLAS_DEFAULT_MATH));
    // CUBLAS_CHECK(hipblasSetMathMode(handle, HIPBLAS_TF32_TENSOR_OP_MATH));

    // Compute ground truth with cuBLAS.
    gemvCublas(thrust::raw_pointer_cast(d_a.data()),
               thrust::raw_pointer_cast(d_x.data()),
               thrust::raw_pointer_cast(d_y.data()),
               m,
               n,
               alpha,
               beta,
               handle);
    golden_y = d_y;

    constexpr int kBlockDimX = 128;
    constexpr int kBlockDimY = 1;
    constexpr int kBlockSpanX = 128;
    constexpr int kBlockSpanY = 1;
    constexpr dim3 kBlock(kBlockDimX, kBlockDimY);
    dim3 grid(std::min(1, m >> 1));

    // GEMV Naive.
    if (kTestGemvNaive)
    {
        gemvNaive<128, 1, kBlockSpanX, kBlockSpanY><<<grid, kBlock>>>(
               thrust::raw_pointer_cast(d_a.data()),
               thrust::raw_pointer_cast(d_x.data()),
               thrust::raw_pointer_cast(d_y.data()),
               m,
               n,
               alpha,
               beta
        );
        CUDA_CHECK_LAST_ERROR();
        CUDA_CHECK(hipDeviceSynchronize());
        checkResult(d_y, golden_y, m);
    }

    // Free cuda resources.
    CUDA_CHECK(hipEventDestroy(ss));
    CUDA_CHECK(hipEventDestroy(ee));
    CUBLAS_CHECK(hipblasDestroy(handle));

    return EXIT_SUCCESS;
}

// Commonly-used Nsight Compute metrics:
// Bank conflicts:
//     l1tex__data_bank_conflicts_pipe_lsu_mem_shared_op_ld.sum
//     l1tex__data_bank_conflicts_pipe_lsu_mem_shared_op_st.sum
// Warp divergence:
//     smsp__thread_inst_executed_per_inst_executed
// L1/L2 cache hit rates:
//     l1tex__t_sector_hit_rate
//     lts__t_sector_hit_rate
// Global load/store throughputs and efficiencies:
//     l1tex__t_bytes_pipe_lsu_mem_global_op_ld.sum.per_second
//     l1tex__t_bytes_pipe_lsu_mem_global_op_st.sum.per_second
//     smsp__sass_average_data_bytes_per_sector_mem_global_op_ld.pct
//     smsp__sass_average_data_bytes_per_sector_mem_global_op_st.pct
// Achieved and therotical occupancy:
//     sm__warps_active.avg.pct_of_peak_sustained_active
//     sm__maximum_warps_per_active_cycle_pct

/*
# Profile bank conflicts:
ncu -k regex:gemmSmem --metrics l1tex__data_bank_conflicts_pipe_lsu_mem_shared_op_ld.sum,l1tex__data_bank_conflicts_pipe_lsu_mem_shared_op_st.sum ./cmake-build-release/demo
*/
