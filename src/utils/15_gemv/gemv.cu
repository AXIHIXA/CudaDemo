#include "hip/hip_runtime.h"
#include <algorithm>
#include <cassert>
#include <chrono>
#include <cmath>
#include <cstdlib>
#include <iostream>
#include <numeric>
#include <random>
#include <vector>

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/equal.h>
#include <thrust/execution_policy.h>
#include <thrust/host_vector.h>

#include "utils/cuda_utils.h"


/// GEMV kernel for fp32 with cuBLAS, calculating y == alpha * (A @ x) + beta * y.
/// Used for ground truth calculation.
/// cuBLAS matrices are in COLUMN-MAJOR, A should be transposed.
/// \param[in] a      shape=(dm, dn)
/// \param[in] x      shape=(dn, 1)
/// \param[in/out] y  shape=(dn, 1)
void gemvCublas(const float * __restrict__ a,
                const float * __restrict__ x,
                float * __restrict__ y,
                int dm,
                int dn,
                float alpha,
                float beta,
                hipblasHandle_t handle)
{
    CUBLAS_CHECK(
        hipblasSgemv(
            handle, HIPBLAS_OP_T,
            dm, dn,
            &alpha,
            a, dn,
            x, 1,
            &beta,
            y, 1
        )
    );
}


/// GEVM kernel for fp32 with cuBLAS, calculating y == alpha * (x @ A) + beta * y
/// Used for ground truth calculation.
/// cuBLAS matrices are in COLUMN-MAJOR, A should be transposed.
/// \param[in] x      shape=(1, dm)
/// \param[in] a      shape=(dm, dn)
/// \param[in/out] y  shape=(1, dn)
void gevmCublas(const float * __restrict__ x,
                const float * __restrict__ a,
                float * __restrict__ y,
                int dm,
                int dn,
                float alpha,
                float beta,
                hipblasHandle_t handle)
{
    CUBLAS_CHECK(
        hipblasSgemv(
            handle, HIPBLAS_OP_N,
            dm, dn,
            &alpha,
            a, dn,
            x, 1,
            &beta,
            y, 1
        )
    );
}


template <typename acc_t, int kWarpThreads = 32>
__device__ acc_t warpReduce(acc_t val)
{
    static_assert(kWarpThreads == 32);

    #pragma unroll
    for (int step = (kWarpThreads >> 1); 0 < step; step >>= 1)
    {
        val += __shfl_xor_sync(0xffffffff, val, step, kWarpThreads);
    }

    return val;
}


template <int kBlockDimX, int kBlockDimY, int kWarpThreads = 32, typename acc_t>
__device__ acc_t blockReduce(acc_t val)
{
    static_assert(kBlockDimX % kWarpThreads == 0 && kBlockDimY == 1 && kWarpThreads == 32);
    constexpr int kWarps = kBlockDimX / kWarpThreads;
    val = warpReduce(val);

    __shared__ acc_t warpAggregate[kWarps];

    const int laneIdx = threadIdx.x % kWarpThreads;
    const int warpIdx = threadIdx.x / kWarpThreads;

    if (warpIdx < kWarps && laneIdx == 0)
    {
        warpAggregate[warpIdx] = val;
    }

    __syncthreads();

    val = 0;

    #pragma unroll
    for (int warp = 0; warp < kWarps; ++warp)
    {
        val += warpAggregate[warp];
    }

    return val;
}


/// alpha * A @ X + beta * Y.
/// Each 1D block computes one element in output.
template <int kBlockDimX, int kBlockDimY>
__global__ void gemvNaive(
        const float * __restrict__ a,
        const float * __restrict__ x,
        float * __restrict__ y,
        int dm,
        int dn,
        float alpha,
        float beta)
{
    // Alignment requirements for float4 vectorized loads/stores.
    // Note that this works only for Debug builds...
    assert(dm % 4 == 0);

    static_assert(kBlockDimY == 1);
    constexpr int kPackSize = 4;
    constexpr int kBlockSpanX = kPackSize * kBlockDimX;

    float4 threadData;

    // Grid Translation to cover all rows.
    for (int gy = blockIdx.y; gy < dm; gy += gridDim.x)
    {
        threadData.x = 0;
        threadData.y = 0;
        threadData.z = 0;
        threadData.w = 0;

        // Block Translation to cover all columns.
        for (int baseX = 0; baseX < dn; baseX += kBlockSpanX)
        {
            const int gx = baseX + threadIdx.x * kPackSize;

            if (gx < dn)
            {
                float4 a4 = *reinterpret_cast<const float4 *>(a + gy * dn + gx);
                float4 x4 = *reinterpret_cast<const float4 *>(x + gx);
                threadData.x += a4.x * x4.x;
                threadData.y += a4.y * x4.y;
                threadData.z += a4.z * x4.z;
                threadData.w += a4.w * x4.w;
            }
        }

        threadData.x = blockReduce<kBlockDimX, kBlockDimY>(threadData.x);
        threadData.y = blockReduce<kBlockDimX, kBlockDimY>(threadData.y);
        threadData.z = blockReduce<kBlockDimX, kBlockDimY>(threadData.z);
        threadData.w = blockReduce<kBlockDimX, kBlockDimY>(threadData.w);

        if (threadIdx.x == 0)
        {
            y[gy] = alpha * (threadData.x + threadData.y + threadData.z + threadData.w) + beta * y[gy];
        }
    }
}


__device__ __forceinline__
float4 fma(const float4 & a, float b, const float4 & c)
{
    float4 ret;
    ret.x = a.x * b + c.x;
    ret.y = a.y * b + c.y;
    ret.z = a.z * b + c.z;
    ret.w = a.w * b + c.w;
    return ret;
}


__device__ __forceinline__
float4 add(const float4 & a, const float4 & b)
{
    float4 ret;
    ret.x = a.x + b.x;
    ret.y = a.y + b.y;
    ret.z = a.z + b.z;
    ret.w = a.w + b.w;
    return ret;
}


/// GEVM
/// X [1, dn] @ A [dn, dm] --> Y [1, dm]
/// X: Q @ K.T; A: V
/// Each 1D thread block accumulates several rows in A (each row of shape [1, dm]).
/// Each thread handles a vectorized pack in one row,
/// and all threads will span multiple rows (strided w.r.t. block).
/// All elements in one row in A multiply with the same element in X.
template <int kBlockDimX, int kThreadsPerValue, typename Vec = float4>
__global__ void gevm(
        const float * __restrict__ x,
        const float * __restrict__ a,
        float * __restrict__ y,
        int dn,
        int dm)
{
    static_assert(std::is_same_v<Vec, float4>);

    constexpr int kVecSize = sizeof(Vec) / sizeof(float);
    constexpr int kBlockSpanY = kBlockDimX / kThreadsPerValue;

    extern __shared__ unsigned char smem[];
    auto rowBuf = reinterpret_cast<float *>(smem);

    const int tid = threadIdx.x;

    // This thread: (x, y) coordinate to load from matrix A.
    const int ay = tid / kThreadsPerValue;
    const int ax = (tid % kThreadsPerValue) * kVecSize;

    Vec out = {};

    // Gird translation.
    // Reduces dn rows into kBlockSpanY rows.
    for (int yy = ay; yy < dn; yy += kBlockSpanY)
    {
        Vec pack = *reinterpret_cast<const Vec *>(&a[yy * dm + ax]);
        float logits = x[yy];
        out = fma(pack, logits, out);
    }

    // Binary row-wise reduction.
    // Now we have kBlockSpanY rows spanned across this block.
    // We reduce them into one row and write back.
    for (int rowsToReduce = kBlockSpanY; 1 < rowsToReduce; rowsToReduce >>= 1)
    {
        const int mid = rowsToReduce >> 1;

        if (mid <= ay && ay < rowsToReduce)
        {
            *reinterpret_cast<Vec *>(&rowBuf[(ay - mid) * dm + ax]) = out;
        }

        __syncthreads();

        if (ay < mid)
        {
            out = add(*reinterpret_cast<Vec *>(&rowBuf[ay * dm + ax]), out);
        }

        __syncthreads();
    }

    // Write back.
    if (ay == 0)
    {
        *reinterpret_cast<Vec *>(&y[ax]) = out;
    }
}


struct alignas(16) half8
{
    half2 h1 = make_half2(0, 0);
    half2 h2 = make_half2(0, 0);
    half2 h3 = make_half2(0, 0);
    half2 h4 = make_half2(0, 0);
};


__device__ __forceinline__
half8 fma(const half8 & a, half b, const half8 & c)
{
    half8 ret;
    half2 b2 = make_half2(b, b);
    // In case __CUDA_NO_HALF2_OPERATORS__ is defined.
    ret.h1 = __hadd2(__hmul2(a.h1, b2), c.h1);
    ret.h2 = __hadd2(__hmul2(a.h2, b2), c.h2);
    ret.h3 = __hadd2(__hmul2(a.h3, b2), c.h3);
    ret.h4 = __hadd2(__hmul2(a.h4, b2), c.h4);
    return ret;
}


__device__ __forceinline__
half8 add(const half8 & a, const half8 & b)
{
    half8 ret;
    ret.h1 = __hadd2(a.h1, b.h1);
    ret.h2 = __hadd2(a.h2, b.h2);
    ret.h3 = __hadd2(a.h3, b.h3);
    ret.h4 = __hadd2(a.h4, b.h4);
    return ret;
}


///// GEVM for FP16.
///// X [1, dn] @ A [dn, dm] --> Y [1, dm]
///// X: Q @ K.T; A: V
///// Each 1D thread block accumulates several rows in A (each row of shape [1, dm]).
///// Each thread handles a vectorized pack in one row,
///// and all threads will span multiple rows (strided w.r.t. block).
///// All elements in one row in A multiply with the same element in X.
template <int kBlockDimX, int kThreadsPerValue, typename Vec = half8>
__global__ void gevm(
        const half * __restrict__ x,
        const half * __restrict__ a,
        half * __restrict__ y,
        int dn,
        int dm)
{
    static_assert(std::is_same_v<Vec, half8>);

    constexpr int kVecSize = sizeof(Vec) / sizeof(half);
    constexpr int kBlockSpanY = kBlockDimX / kThreadsPerValue;

    extern __shared__ unsigned char smem[];
    auto rowBuf = reinterpret_cast<half *>(smem);

    const int tid = threadIdx.x;

    // This thread: (x, y) coordinate to load from matrix A.
    const int ay = tid / kThreadsPerValue;
    const int ax = (tid % kThreadsPerValue) * kVecSize;

    half8 out = {};

    // Gird translation.
    // Reduces dn rows into kBlockSpanY rows.
    for (int yy = ay; yy < dn; yy += kBlockSpanY)
    {
        half8 pack = *reinterpret_cast<const half8 *>(&a[yy * dm + ax]);
        half logits = x[yy];
        out = fma(pack, logits, out);
    }

    // Binary row-wise reduction.
    // Now we have kBlockSpanY rows spanned across this block.
    // We reduce them into one row and write back.
    for (int rowsToReduce = kBlockSpanY; 1 < rowsToReduce; rowsToReduce >>= 1)
    {
        const int mid = rowsToReduce >> 1;

        if (mid <= ay && ay < rowsToReduce)
        {
            *reinterpret_cast<Vec *>(&rowBuf[(ay - mid) * dm + ax]) = out;
        }

        __syncthreads();

        if (ay < mid)
        {
            out = add(*reinterpret_cast<Vec *>(&rowBuf[ay * dm + ax]), out);
        }

        __syncthreads();
    }

    // Write back.
    if (ay == 0)
    {
        *reinterpret_cast<Vec *>(&y[ax]) = out;
    }
}


template <typename T>
struct Equal
{
    __host__ __device__
    inline bool operator()(const T & a, const T & b) = delete;
};


template <>
struct Equal<float>
{
    __host__ __device__
    inline bool operator()(float a, float b)
    {
        return abs(a - b) < kAbsTol + kRelTol * abs(b);
    }

    static constexpr float kAbsTol = 1e-3f;
    static constexpr float kRelTol = 2e-4f;
};


template <bool kDebugOutput = true, typename acc_t>
void checkResult(const thrust::device_vector<acc_t> & result,
                 const thrust::device_vector<acc_t> & golden,
                 int dm)
{
    bool resultIsCorrect = thrust::equal(thrust::device, result.cbegin(), result.cend(), golden.cbegin(), Equal<acc_t>());

    if constexpr (kDebugOutput)
    {
        if (!resultIsCorrect)
        {
            thrust::host_vector<acc_t> a = result;
            thrust::host_vector<acc_t> b = golden;

            printf("Result:\n");
            for (int i = 0; i < dm; ++i)
            {
                printf("%6.2f ", a[i]);
            }
            printf("\n\n");

            printf("Ground truth:\n");
            for (int i = 0; i < dm; ++i)
            {
                printf("%6.2f ", b[i]);
            }
            printf("\n\n");
        }
    }

    std::printf("Result is %s\n\n", resultIsCorrect ? "correct." : "WRONG!!!");
}


void displayInputs(const thrust::host_vector<float> & h_a,
                   const thrust::host_vector<float> & h_x,
                   int m,
                   int n,
                   float alpha,
                   float beta)
{
    printf("alpha = %f, beta = %f\n", alpha, beta);

    printf("A:\n");
    for (int y = 0; y < m; ++y)
    {
        for (int x = 0; x < n; ++x)
        {
            printf("%6.0f ", h_a[y * n + x]);
        }
        printf("\n");
    }

    printf("X:\n");
    for (int y = 0; y < m; ++y)
    {
        printf("%6.0f ", h_x[y]);
    }
    printf("\n");
}


int main(int argc, char * argv[])
{
    /// Switches for debugging output correctness.
    /// \param kDup        Set to 1 to debug output (kernel only launched once) and results will be checked.
    ///                    Set to values greater than 1 to profile.
    ///                    In the latter case, results will NOT be checked because it's in-place GEMM.
    ///                    We do not dispatch by build type because we have -G flag for Debug builds
    ///                    (that's for debugging runtime errors).
    /// \param kRandInput  Whether we random input matrices.
    ///                    Enable when checking correctness or profiling.
    ///                    Disable when debugging output.
    constexpr int kDup = 1;
    constexpr bool kRandInput = false;

    constexpr bool kTestGemvNaive = false;
    constexpr bool kTestGevmFp32 = false;
    constexpr bool kTestGevmFp16 = true;

    // Problem setting.
    const int problemSize = 1024;
    const int m = problemSize;
    const int n = problemSize;
    float alpha = 1.0f;
    float beta = 0.0f;
    thrust::host_vector<float> h_a(m * n, 1.0f);
    thrust::host_vector<float> h_x(m, 1.0f);
    thrust::host_vector<float> h_y(std::max(m, n), 0.0f);
//    std::iota(h_a.begin(), h_a.end(), 1.0f);
//    std::iota(h_x.begin(), h_x.end(), 1.0f);

    if constexpr (kRandInput)
    {
        unsigned seed = std::random_device()();
        // std::printf("seed = %u\n", seed);
        std::default_random_engine e(seed);
        // std::normal_distribution<float> d(0.0f, 1.0f);
        std::uniform_int_distribution d(1, 20);
        auto g = [&e, &d]() -> float { return d(e); };
        alpha = g();
        beta = g();
        std::generate(h_a.begin(), h_a.end(), g);
        std::generate(h_x.begin(), h_x.end(), g);
        std::generate(h_y.begin(), h_y.end(), g);
    }

    // displayInputs(h_a, h_x, m, n, alpha, beta);

    thrust::device_vector<float> golden_y(n);
    thrust::device_vector<float> d_a = h_a;
    thrust::device_vector<float> d_x = h_x;
    thrust::device_vector<float> d_y = h_y;

    // CUDA resources that require manual destruction.
    float ms;
    hipEvent_t ss, ee;
    CUDA_CHECK(hipEventCreate(&ss));
    CUDA_CHECK(hipEventCreate(&ee));

    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    // Testing says that these two modes are the same.
    CUBLAS_CHECK(hipblasSetMathMode(handle, HIPBLAS_DEFAULT_MATH));
    // CUBLAS_CHECK(hipblasSetMathMode(handle, HIPBLAS_TF32_TENSOR_OP_MATH));

    // GEMV Naive.
    if (kTestGemvNaive)
    {
        // Compute GEMV ground truth with cuBLAS.
        gemvCublas(thrust::raw_pointer_cast(d_a.data()),
                   thrust::raw_pointer_cast(d_x.data()),
                   thrust::raw_pointer_cast(d_y.data()),
                   m,
                   n,
                   alpha,
                   beta,
                   handle);
        golden_y = d_y;

        // GEMV
        d_y = h_y;

        constexpr int kBlockDimX = 128;
        constexpr int kBlockDimY = 1;
        constexpr dim3 kBlock(kBlockDimX, kBlockDimY);
        dim3 grid(std::min(1, m >> 1));

        gemvNaive<kBlockDimX, kBlockDimY><<<grid, kBlock>>>(
               thrust::raw_pointer_cast(d_a.data()),
               thrust::raw_pointer_cast(d_x.data()),
               thrust::raw_pointer_cast(d_y.data()),
               m,
               n,
               alpha,
               beta
        );
        CUDA_CHECK_LAST_ERROR();
        CUDA_CHECK(hipDeviceSynchronize());

        checkResult(d_y, golden_y, m);
    }

    if constexpr (kTestGevmFp32)
    {
        // Compute GEMV ground truth with cuBLAS.
        gemvCublas(thrust::raw_pointer_cast(d_a.data()),
                   thrust::raw_pointer_cast(d_x.data()),
                   thrust::raw_pointer_cast(d_y.data()),
                   m,
                   n,
                   alpha,
                   beta,
                   handle);

        golden_y = d_y;

        // GEVM
        d_y = h_y;

        constexpr int kBlockDimX = 256;
        using T = float;
        using Vec = float4;
        constexpr int kThreadsPerValue = n * sizeof(T) / sizeof(Vec);
        static_assert(kBlockDimX % kThreadsPerValue == 0);
        const int smemBytes = (1 + kBlockDimX / kThreadsPerValue) * n * sizeof(T);

        gevm<kBlockDimX, kThreadsPerValue, Vec><<<1, kBlockDimX, smemBytes>>>(
                thrust::raw_pointer_cast(d_x.data()),
                thrust::raw_pointer_cast(d_a.data()),
                thrust::raw_pointer_cast(d_y.data()),
                m,
                n
        );

        CUDA_CHECK_LAST_ERROR();
        CUDA_CHECK(hipDeviceSynchronize());

        std::printf("FP32 GEVM ");
        checkResult(d_y, golden_y, n);
    }

    if constexpr (kTestGevmFp16)
    {
        // GEVM
        thrust::device_vector<half> d_x(1 * m, static_cast<half>(1));
        thrust::device_vector<half> d_a(m * n, static_cast<half>(1));
        thrust::device_vector<half> d_y(1 * n, static_cast<half>(0));

        constexpr int kBlockDimX = 256;
        using T = half;
        using Vec = half8;
        constexpr int kThreadsPerValue = n * sizeof(T) / sizeof(Vec);
        static_assert(kBlockDimX % kThreadsPerValue == 0);
        const int smemBytes = (1 + kBlockDimX / kThreadsPerValue) * n * sizeof(T);

        gevm<kBlockDimX, kThreadsPerValue, Vec><<<1, kBlockDimX, smemBytes>>>(
                thrust::raw_pointer_cast(d_x.data()),
                thrust::raw_pointer_cast(d_a.data()),
                thrust::raw_pointer_cast(d_y.data()),
                m,
                n
        );

        CUDA_CHECK_LAST_ERROR();
        CUDA_CHECK(hipDeviceSynchronize());

        thrust::host_vector<half> h_x = d_x;

        for (int i = 0; i < n; ++i)
        {
            std::printf("%6.2f ", static_cast<float>(h_x[i]));
        }

        std::printf("\n");
    }

    // Free cuda resources.
    CUDA_CHECK(hipEventDestroy(ss));
    CUDA_CHECK(hipEventDestroy(ee));
    CUBLAS_CHECK(hipblasDestroy(handle));

    return EXIT_SUCCESS;
}

// Commonly-used Nsight Compute metrics:
// Bank conflicts:
//     l1tex__data_bank_conflicts_pipe_lsu_mem_shared_op_ld.sum
//     l1tex__data_bank_conflicts_pipe_lsu_mem_shared_op_st.sum
// Warp divergence:
//     smsp__thread_inst_executed_per_inst_executed
// L1/L2 cache hit rates:
//     l1tex__t_sector_hit_rate
//     lts__t_sector_hit_rate
// Global load/store throughputs and efficiencies:
//     l1tex__t_bytes_pipe_lsu_mem_global_op_ld.sum.per_second
//     l1tex__t_bytes_pipe_lsu_mem_global_op_st.sum.per_second
//     smsp__sass_average_data_bytes_per_sector_mem_global_op_ld.pct
//     smsp__sass_average_data_bytes_per_sector_mem_global_op_st.pct
// Achieved and therotical occupancy:
//     sm__warps_active.avg.pct_of_peak_sustained_active
//     sm__maximum_warps_per_active_cycle_pct

/*
# Profile bank conflicts:
ncu -k regex:gemmSmem --metrics l1tex__data_bank_conflicts_pipe_lsu_mem_shared_op_ld.sum,l1tex__data_bank_conflicts_pipe_lsu_mem_shared_op_st.sum ./cmake-build-release/demo
*/


/*
Compute sanitizer to check memory access:
compute-sanitizer --tool memcheck ./cmake-build-debug/demo
*/