#define TEST_REDUCE_V5
//#define TEST_REDUCE_V6

#ifdef TEST_REDUCE_V5
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>


#define THREAD_PER_BLOCK 256
// latency: 0.656ms
// v5：循环展开
template <int blockSize>
__device__ void BlockSharedMemReduce(float* smem) {
    //对v4 L45的for循环展开，以减去for循环中的加法指令，以及给编译器更多重排指令的空间
  if (blockSize >= 1024) {
    if (threadIdx.x < 512) {
      smem[threadIdx.x] += smem[threadIdx.x + 512];
    }
    __syncthreads();
  }
  if (blockSize >= 512) {
    if (threadIdx.x < 256) {
      smem[threadIdx.x] += smem[threadIdx.x + 256];
    }
    __syncthreads();
  }
  if (blockSize >= 256) {
    if (threadIdx.x < 128) {
      smem[threadIdx.x] += smem[threadIdx.x + 128];
    }
    __syncthreads();
  }
  if (blockSize >= 128) {
    if (threadIdx.x < 64) {
      smem[threadIdx.x] += smem[threadIdx.x + 64];
    }
    __syncthreads();
  }
  // the final warp
  if (threadIdx.x < 32) {
    volatile float* vshm = smem;
    if (blockDim.x >= 64) {
      vshm[threadIdx.x] += vshm[threadIdx.x + 32];
    }
    vshm[threadIdx.x] += vshm[threadIdx.x + 16];
    vshm[threadIdx.x] += vshm[threadIdx.x + 8];
    vshm[threadIdx.x] += vshm[threadIdx.x + 4];
    vshm[threadIdx.x] += vshm[threadIdx.x + 2];
    vshm[threadIdx.x] += vshm[threadIdx.x + 1];
  }
}

template <int blockSize>
__global__ void reduce_v5(float *d_in, float *d_out){
    __shared__ float smem[THREAD_PER_BLOCK];
    // 泛指当前线程在其block内的id
    unsigned int tid = threadIdx.x;
    // 泛指当前线程在所有block范围内的全局id, *2代表当前block要处理2*blocksize的数据
    // ep. blocksize = 2, blockIdx.x = 1, when tid = 0, gtid = 4, gtid + blockSize = 6; when tid = 1, gtid = 5, gtid + blockSize = 7
    // ep. blocksize = 2, blockIdx.x = 0, when tid = 0, gtid = 0, gtid + blockSize = 2; when tid = 1, gtid = 1, gtid + blockSize = 3
    // so, we can understand L59, one thread handle data located in tid and tid + blockSize
    unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
    // load: 每个线程加载两个元素到shared mem对应位置
    smem[tid] = d_in[i] + d_in[i + blockDim.x];
    __syncthreads();
    // compute: reduce in shared mem
    BlockSharedMemReduce<blockSize>(smem);

    // store: 哪里来回哪里去，把reduce结果写回显存
    // GridSize个block内部的reduce sum已得出，保存到d_out的每个索引位置
    if (tid == 0) {
        d_out[blockIdx.x] = smem[0];
    }
}

bool CheckResult(float *out, float groudtruth, int n){
    float res = 0;
    for (int i = 0; i < n; i++){
        res += out[i];
    }
    if (res != groudtruth) {
        return false;
    }
    return true;
}

int main(){
    float milliseconds = 0;

    const int N = 25600000;
    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    const int blockSize = 256;
    int GridSize = std::min((N + 256 - 1) / 256, deviceProp.maxGridSize[0]);
    //int GridSize = 100000;
    float *a = (float *)malloc(N * sizeof(float));
    float *d_a;
    hipMalloc((void **)&d_a, N * sizeof(float));

    float *out = (float*)malloc((GridSize) * sizeof(float));
    float *d_out;
    hipMalloc((void **)&d_out, (GridSize) * sizeof(float));

    for(int i = 0; i < N; i++){
        a[i] = 1.0f;
    }

    float groudtruth = N * 1.0f;

    hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 Grid(GridSize);
    dim3 Block(blockSize / 2);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    for (int i = 0; i < 100; ++i)
    reduce_v5<blockSize / 2><<<Grid,Block>>>(d_a, d_out);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(out, d_out, GridSize * sizeof(float), hipMemcpyDeviceToHost);
    printf("allcated %d blocks, data counts are %d \n", GridSize, N);
    bool is_right = CheckResult(out, groudtruth, GridSize);
    if(is_right) {
        printf("the ans is right\n");
    } else {
        printf("the ans is wrong\n");
        for(int i = 0; i < GridSize;i++){
            printf("resPerBlock : %lf ",out[i]);
        }
        printf("\n");
        printf("groudtruth is: %f \n", groudtruth);
    }
    printf("reduce_v5 latency = %f ms\n", milliseconds / 100.0f);

    hipFree(d_a);
    hipFree(d_out);
    free(a);
    free(out);
}
#endif  // TEST_REDUCE_V5

#ifdef TEST_REDUCE_V6
#include <bits/stdc++.h>
#include <cuda.h>
#include "cuda_runtime.h"
// 注意: v0-v5里面kernel得到的是各个block负责范围内的总和，要想得到最终的和，需要把各个block求得的总和再做reduce sum
// v6: multi-block reduce final result by two pass
// latency: 1.815ms
template <int blockSize>
__device__ void BlockSharedMemReduce(float* smem) {
    //对v4 L45的for循环展开，以减去for循环中的加法指令，以及给编译器更多重排指令的空间
  if (blockSize >= 1024) {
    if (threadIdx.x < 512) {
      smem[threadIdx.x] += smem[threadIdx.x + 512];
    }
    __syncthreads();
  }
  if (blockSize >= 512) {
    if (threadIdx.x < 256) {
      smem[threadIdx.x] += smem[threadIdx.x + 256];
    }
    __syncthreads();
  }
  if (blockSize >= 256) {
    if (threadIdx.x < 128) {
      smem[threadIdx.x] += smem[threadIdx.x + 128];
    }
    __syncthreads();
  }
  if (blockSize >= 128) {
    if (threadIdx.x < 64) {
      smem[threadIdx.x] += smem[threadIdx.x + 64];
    }
    __syncthreads();
  }
  // the final warp
  if (threadIdx.x < 32) {
    volatile float* vshm = smem;
    if (blockDim.x >= 64) {
      vshm[threadIdx.x] += vshm[threadIdx.x + 32];
    }
    vshm[threadIdx.x] += vshm[threadIdx.x + 16];
    vshm[threadIdx.x] += vshm[threadIdx.x + 8];
    vshm[threadIdx.x] += vshm[threadIdx.x + 4];
    vshm[threadIdx.x] += vshm[threadIdx.x + 2];                                                                                                                                                                                          vshm[threadIdx.x] += vshm[threadIdx.x + 1];

  }
}

template <int blockSize>
__global__ void reduce_v6(float *d_in, float *d_out, int nums){
    __shared__ float smem[blockSize];
    // 泛指当前线程在其block内的id
    unsigned int tid = threadIdx.x;
    // 泛指当前线程在所有block范围内的全局id
    unsigned int gtid = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int total_thread_num = blockDim.x * gridDim.x;
    // 基于v5的改进：不用显式指定一个线程处理2个元素，而是通过L58的for循环来自动确定每个线程处理的元素个数
    float sum = 0.0f;
    for (int32_t i = gtid; i < nums; i += total_thread_num) {
        sum += d_in[i];
    }
    smem[tid] = sum;
    __syncthreads();
    // compute: reduce in shared mem
    BlockSharedMemReduce<blockSize>(smem);

    // store: 哪里来回哪里去，把reduce结果写回显存
    // GridSize个block内部的reduce sum已得出，保存到d_out的每个索引位置
    if (tid == 0) {
        d_out[blockIdx.x] = smem[0];
    }
}

bool CheckResult(float *out, float groudtruth, int n){
    if (*out != groudtruth) {
      return false;
    }
    return true;
}

int main(){
    cudaDeviceProp deviceProp;
    cudaGetDeviceProperties(&deviceProp, 0);
    int maxblocks = deviceProp.maxGridSize[0];
    const int blockSize = 256;
    const int N = 25600000;
    int gridSize = std::min((N + blockSize - 1) / blockSize, maxblocks);

    float milliseconds = 0;
    float *a = (float *)malloc(N * sizeof(float));
    float *d_a;
    cudaMalloc((void **)&d_a,N * sizeof(float));

    float *out = (float*)malloc((gridSize) * sizeof(float));
    float *d_out;
    float *part_out;//新增part_out存储每个block reduce的结果
    cudaMalloc((void **)&d_out, 1 * sizeof(float));
    cudaMalloc((void **)&part_out, (gridSize) * sizeof(float));
    float groudtruth = N;

    for(int i = 0; i < N; i++){
        a[i] = 1;
    }

    cudaMemcpy(d_a, a, N * sizeof(float), cudaMemcpyHostToDevice);

    dim3 Grid(gridSize);
    dim3 Block(blockSize);


    cudaEvent_t start, stop;
    cudaEventCreate(&start);
    cudaEventCreate(&stop);
    cudaEventRecord(start);
    for (int i = 0; i < 100; ++i) {
    reduce_v6<blockSize><<<Grid, Block>>>(d_a, part_out, N);
    reduce_v6<blockSize><<<1, Block>>>(part_out, d_out, gridSize); }
    cudaEventRecord(stop);
    cudaEventSynchronize(stop);
    cudaEventElapsedTime(&milliseconds, start, stop);

    cudaMemcpy(out, d_out, 1 * sizeof(float), cudaMemcpyDeviceToHost);
    bool is_right = CheckResult(out, groudtruth, 1);
    if(is_right) {
        printf("the ans is right\n");
    } else {
        printf("the ans is wrong\n");
        for(int i = 0;i < 1;i++){
            printf("%lf ",out[i]);
        }
        printf("\n");
    }
    printf("reduce_v6 latency = %f ms\n", milliseconds / 100.0f);

    cudaFree(d_a);
    cudaFree(d_out);
    cudaFree(part_out);
    free(a);
    free(out);
}
#endif  // TEST_REDUCE_V6
