#include "hip/hip_runtime.h"
#include <algorithm>
#include <chrono>
#include <cmath>
#include <cstdlib>
#include <iostream>
#include <numeric>
#include <vector>

#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "utils/cuda_utils.h"


template <typename T>
__global__ void transposeNaive(const T * __restrict__ in, int nx, int ny, T * __restrict__ out)
{
    unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < nx && y < ny)
    {
        unsigned from = y * nx + x;
        unsigned to = x * ny + y;
        out[to] = in[from];
    }
}


/// Read input in row-major, store into smem (padded) in row-major.
/// Write to output in row major.
/// This will essentially read smem in column-major.
/// That is, a thread will NOT write the data it reads.
///
/// Do NOT exit if thread id is out of bound!
/// Because threads do not write what they read,
/// margin (remainder) elements will be read by in-bound threads
/// (with these in-bound threads write NOTHING)
/// but written by out-of-bound threads!
template <int kPadding = 1, typename T>
__global__ void transpose(const T * __restrict__ in, int nx, int ny, T * __restrict__ out)
{
    extern __shared__ T smem[];  // smem[blockDim.y][blockDim.x + kPadding].

    unsigned ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned iy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned from = iy * nx + ix;
    unsigned s = threadIdx.y * (blockDim.x + kPadding) + threadIdx.x;

    if (ix < nx && iy < ny)
    {
        smem[s] = in[from];
    }

    __syncthreads();

    unsigned t = threadIdx.y * blockDim.x + threadIdx.x;

    // This thread will write t-th (column major) element in smem.
    // What's its x, y coordinates in this block?
    unsigned sx = t / blockDim.y;
    unsigned sy = t % blockDim.y;
    s = sy * (blockDim.x + kPadding) + sx;

    ix = blockIdx.y * blockDim.y + sy;
    iy = blockIdx.x * blockDim.x + sx;
    unsigned to = iy * ny + ix;

    if (ix < ny && iy < nx)
    {
        out[to] = smem[s];
    }
}


/// WARNING: Unrolling does not guarantee better performace! Performance may degrade!
/// "Row unroll":
/// Each thread block is "duplicated" for kNItems times along the x-dimension.
/// Threads first do what they do in the "original block",
/// then they translate/slide to the next block and do what they do in that area.
/// The x-coordinate offsets by blockDim.x.
///
/// E.g.
/// [[ 0  1  2  3  4  5  6  7],
///  [ 8  9 10 11 12 13 14 15]].
/// blockDim == {2, 2}, kNItems == 2.
/// Threads in block #0 first load [0, 1, 8, 9] into smem,
/// then load [2, 3, 9, 10].
/// For sequential loads in for loop, each thread simply offset x coordinate by blockDim.x.
/// Smem index also offsets by blockDim.x. (The padding does not affect here as we don't cross rows.)
template <int kNItems = 1, int kPadding = 1, typename T>
__global__ void transposeUnroll(const T * __restrict__ in, int nx, int ny, T * __restrict__ out)
{
    extern __shared__ T smem[];

    unsigned ix = blockIdx.x * blockDim.x * kNItems + threadIdx.x;
    unsigned iy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned from = iy * nx + ix;
    unsigned s = threadIdx.y * (blockDim.x * kNItems + kPadding) + threadIdx.x;

    // #pragma unroll requires trip count be constant!
    // https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#pragma-unroll
    #pragma unroll
    for (int u = 0; u < kNItems; ++u)
    {
        if (ix + blockDim.x * u < nx && iy < ny)
        {
            smem[s + blockDim.x * u] = in[from + blockDim.x * u];
        }
    }

    __syncthreads();

    // t is the INDEX of this thread in block, not index of its first element!
    unsigned t = threadIdx.y * blockDim.x + threadIdx.x;
    unsigned sx = t / blockDim.y;
    unsigned sy = t % blockDim.x;
    s = sy * (blockDim.x * kNItems + kPadding) + sx;

    ix = blockIdx.y * blockDim.y + sy;
    iy = blockIdx.x * blockDim.x * kNItems + sx;
    unsigned to = iy * ny + ix;

    #pragma unroll
    for (int u = 0; u < kNItems; ++u)
    {
        if (ix < ny && iy + blockDim.x * u < nx)
        {
            out[to + ny * blockDim.x * u] = smem[s + blockDim.x * u];
        }
    }
}


void print(const thrust::host_vector<float> & in, int r, int c, const thrust::host_vector<float> & out)
{
    for (int i = 0; i < r; ++i)
    {
        for (int j = 0; j < c; ++j)
        {
            printf("%.1f ", in[i * c + j]);
        }

        std::printf("\n");
    }

    std::printf("\n");

    for (int i = 0; i < c; ++i)
    {
        for (int j = 0; j < r; ++j)
        {
            printf("%.1f ", out[i * r + j]);
        }

        std::printf("\n");
    }

    std::printf("\n");
}


void checkResult(const thrust::host_vector<float> & in, int r, int c, const thrust::host_vector<float> & out)
{
    #ifndef NDEBUG
    print(in, r, c, out);
    #endif

    bool resultIsCorrect = true;

    for (int i = 0, shouldBreak = false; !shouldBreak && i < r; ++i)
    {
        for (int j = 0; !shouldBreak && j < c; ++j)
        {
            if (in[i * c + j] != out[j * r + i])
            {
                resultIsCorrect = false;
                shouldBreak = true;
            }
        }
    }

    std::printf("Result is %s\n\n", resultIsCorrect ? "correct." : "WRONG!!!");
}


int main(int argc, char * argv[])
{
    constexpr int r = 4096;
    constexpr int c = 2048;
    constexpr int rc = r * c;
    thrust::host_vector<float> h_in(rc);
    std::iota(h_in.begin(), h_in.end(), 0.0f);
    thrust::device_vector<float> d_in = h_in;
    thrust::device_vector<float> d_out(rc);
    thrust::host_vector<float> h_out;

    constexpr dim3 block = {32, 32};
    dim3 grid = {(c + block.x - 1) / block.x, (r + block.y - 1) / block.y};

    constexpr int kPad = 1;
    constexpr int kNItems = 2;
    constexpr int kDup = 100;

    using Clock = std::chrono::high_resolution_clock;
    Clock::time_point ss, ee;

    // Naive
    thrust::fill(d_out.begin(), d_out.end(), 0.0f);
    transposeNaive<<<grid, block>>>(d_in.data().get(), c, r, d_out.data().get());
    CUDA_CHECK(hipDeviceSynchronize());
    h_out = d_out;
    std::printf("transposeNaive: ");
    checkResult(h_in, r, c, h_out);

    // Regular
    thrust::fill(thrust::device, d_out.begin(), d_out.end(), 0.0f);
    transpose<kPad><<<grid, block, (block.x + kPad) * block.y * sizeof(float)>>>(
            d_in.data().get(), c, r, d_out.data().get());
    CUDA_CHECK(hipDeviceSynchronize());
    ss = Clock::now();
    for (int dup = 0; dup < kDup; ++dup)
    {
        transpose<kPad><<<grid, block, (block.x + kPad) * block.y * sizeof(float)>>>(
            d_in.data().get(), c, r, d_out.data().get());
    }
    CUDA_CHECK(hipDeviceSynchronize());
    ee = Clock::now();
    h_out = d_out;
    std::printf("transpose: ");
    std::printf("took %f ms, ", std::chrono::duration_cast<std::chrono::microseconds>(ee - ss).count() * 1e-3f);
    checkResult(h_in, r, c, h_out);

    // Unroll
    thrust::fill(thrust::device, d_out.begin(), d_out.end(), 0.0f);
    transposeUnroll<kNItems, kPad><<<grid, block, (block.x * kNItems + kPad) * block.y * sizeof(float)>>>(
                d_in.data().get(), c, r, d_out.data().get());
    CUDA_CHECK(hipDeviceSynchronize());
    ss = Clock::now();
    for (int dup = 0; dup < kDup; ++dup)
    {
        transposeUnroll<kNItems, kPad><<<grid, block, (block.x * kNItems + kPad) * block.y * sizeof(float)>>>(
                d_in.data().get(), c, r, d_out.data().get());
    }
    CUDA_CHECK(hipDeviceSynchronize());
    ee = Clock::now();
    h_out = d_out;
    std::printf("transposeUnroll: ");
    std::printf("took %f ms, ", std::chrono::duration_cast<std::chrono::microseconds>(ee - ss).count() * 1e-3f);
    checkResult(h_in, r, c, h_out);

    return EXIT_SUCCESS;
}

/*
# Profile gld_throughput, gld_efficiency, gst_throughput and gst_efficiency.
ncu -k regex:transpose --metrics \
l1tex__t_bytes_pipe_lsu_mem_global_op_ld.sum.per_second,\
l1tex__t_bytes_pipe_lsu_mem_global_op_st.sum.per_second,\
smsp__sass_average_data_bytes_per_sector_mem_global_op_ld.pct,\
smsp__sass_average_data_bytes_per_sector_mem_global_op_st.pct \
./cmake-build-release/demo

# Profile all common metrics.
ncu -k regex:transpose ./cmake-build-release/demo

# For runtime profiling.
nvprof ./cmake-build-release/demo
*/