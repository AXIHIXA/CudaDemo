#include "hip/hip_runtime.h"
#include <algorithm>
#include <chrono>
#include <cmath>
#include <cstdlib>
#include <iostream>
#include <numeric>
#include <random>
#include <vector>

#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "utils/cuda_utils.h"


/// Softmax on innermost dimension.
void cpuSoftmax(const float * __restrict__ in, float * __restrict__ out, int nx, int ny)
{
    for (int y = 0; y < ny; ++y)
    {
        float rowSum = 0.0f;
        float rowMax = std::numeric_limits<float>::min();

        for (int x = 0; x < nx; ++x)
        {
            rowMax = std::max(in[y * nx + x], rowMax);
        }

        for (int x = 0; x < nx; ++x)
        {
            rowSum += std::exp(in[y * nx + x] - rowMax);
        }

        for (int x = 0; x < nx; ++x)
        {
            out[y * nx + x] = std::exp(in[y * nx + x] - rowMax) / rowSum;
        }
    }
}


template <typename T>
struct Max
{
    __device__ __forceinline__ constexpr bool operator()(const T & a, const T & b)
    {
        return (a < b) ? b : a;
    }
};


template <typename T>
struct Sum
{
    __device__ __forceinline__ T operator()(const T & a, const T & b)
    {
        return a + b;
    }
};


/// Bufferfly warp reduction.
/// See https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#reduction-across-a-warp
template <template <typename> class ReductionOp, typename T, int kWarpThreads = 32>
__inline__ __device__ T warpReduce(T val)
{
    #pragma unroll
    for (int mask = kWarpThreads >> 1; 0 < mask; mask >>= 1)
    {
        val = ReductionOp<T>()(val, __shfl_xor_sync(0xffffffff, val, mask, kWarpThreads));
    }

    return val;
}


// Used for vectorized stores and loads.
template <typename T, int kSize>
struct alignas(sizeof(T) * kSize) Vec
{
    T val[kSize];
};


template <int kPackSize>
__device__ void vecLdSt(const float * src, float * dst)
{
    using Vec = Vec<float, kPackSize>;
    *reinterpret_cast<Vec *>(dst) = *(reinterpret_cast<const Vec *>(src));
}


/// Each warp handles a consecutive of kThreadSpanY rows in src.
/// Each row in thread block constitutes a warp.
/// Grid is 1D, spans in y direction.
template <int kBlockDimX = 32, int kBlockDimY, int kThreadSpanX, int kThreadSpanY, int kPackSize, int kWarpThreads = 32>
__global__ void softmax(const float * __restrict__ src,
                        float * __restrict__ dst,
                        int nx,
                        int ny)
{
    // Alignment requirements for vectorized loads/stores.
    // Each warp must be long enough to cover a row.
    // Grid must be 1D and span by the y dimension.
    if (kThreadSpanX * kWarpThreads < nx || nx % kPackSize != 0 || gridDim.x != 1 || gridDim.z != 1)
    {
        __trap();
    }

    // Each warp handles a complete row of input.
    // Each warp must consist of a complete row of threads in the thread block.
    static_assert(kWarpThreads == 32 && kBlockDimX == kWarpThreads);

    // Number of packs (vectorized load granularity on x dimension) should be integer.
    static_assert(kThreadSpanX % kPackSize == 0);

    constexpr float kMinusInfinity = -10000000000.0f;
    constexpr int kNumPacks = kThreadSpanX / kPackSize;

    const int tid = threadIdx.y * kBlockDimX + threadIdx.x;
    const int laneIdx = tid % kWarpThreads;
    const int globalWarpIdx = blockIdx.y * kBlockDimY + threadIdx.y;

    // Register.
    float buf[kThreadSpanY][kThreadSpanX];

    // Grid translation.
    for (int baseY = globalWarpIdx * kThreadSpanY; baseY < ny; baseY += gridDim.y * kBlockDimY * kThreadSpanY)
    {
        float threadMax[kThreadSpanY];

        // Rows handled by a warp.
        // Load into register and reduce thread max.
        #pragma unroll
        for (int rowIdx = 0; rowIdx < kThreadSpanY; ++rowIdx)
        {
            const int gy = baseY + rowIdx;

            // Must be outside, other threadMax might have uninitalized values!
            float * rowBuf = buf[rowIdx];
            threadMax[rowIdx] = kMinusInfinity;

            if (gy < ny)
            {
                // Vectorized load packs in this warp (row), by this thread.
                // Each threads loads kThreadSpanX elements into its rowBuf (GMEM -> REG).
                // Loads are done by kNumPacks vectorized packs, each pack of length kPackSize.
                // Packs by the same threads are strided, so that warp GMEM loads are coalesced.
                #pragma unroll
                for (int packIdx = 0; packIdx < kNumPacks; ++packIdx)
                {
                    // p0 p0 p0  p1 p1 p1  p2 p2 p2
                    const int gx = packIdx * kPackSize * kWarpThreads + laneIdx * kPackSize;
                    const int packOffset = packIdx * kPackSize;

                    if (gx < nx)
                    {
                        vecLdSt<kPackSize>(src + gy * nx + gx, rowBuf + packOffset);

                        #pragma unroll
                        for (int pi = 0; pi < kPackSize; ++pi)
                        {
                            threadMax[rowIdx] = max(threadMax[rowIdx], rowBuf[packOffset + pi]);
                        }
                    }
                    else
                    {
                        #pragma unroll
                        for (int pi = 0; pi < kPackSize; ++pi)
                        {
                            rowBuf[packOffset + pi] = kMinusInfinity;
                        }
                    }
                }
            }
        }

        // Reduce per-warp row max.
        float rowMax[kThreadSpanY];

        #pragma unroll
        for (int rowIdx = 0; rowIdx < kThreadSpanY; ++rowIdx)
        {
            rowMax[rowIdx] = warpReduce<Max, float>(threadMax[rowIdx]);
        }

        // In-place exp(x - rowmax), accumulate thread sum.
        float threadSum[kThreadSpanY];

        #pragma unroll
        for (int rowIdx = 0; rowIdx < kThreadSpanY; ++rowIdx)
        {
            float * rowBuf = buf[rowIdx];
            threadSum[rowIdx] = 0;

            #pragma unroll
            for (int xi = 0; xi < kThreadSpanX; ++xi)
            {
                rowBuf[xi] = exp(rowBuf[xi] - rowMax[rowIdx]);
                threadSum[rowIdx] += rowBuf[xi];
            }
        }

        // Reduce per-warp row sum.
        float rowSum[kThreadSpanY];

        #pragma unroll
        for (int rowIdx = 0; rowIdx < kThreadSpanY; ++rowIdx)
        {
            rowSum[rowIdx] = warpReduce<Sum, float>(threadSum[rowIdx]);
        }

        // In-place softmax division by rowSum and write-back.
        #pragma unroll
        for (int rowIdx = 0; rowIdx < kThreadSpanY; ++rowIdx)
        {
            const int gy = baseY + rowIdx;

            if (gy < ny)
            {
                float * rowBuf = buf[rowIdx];

                #pragma unroll
                for (int xi = 0; xi < kThreadSpanX; ++xi)
                {
                    rowBuf[xi] /= rowSum[rowIdx];
                }

                #pragma unroll
                for (int packIdx = 0; packIdx < kNumPacks; ++packIdx)
                {
                    // p0 p0 p0  p1 p1 p1  p2 p2 p2
                    const int gx = packIdx * kPackSize * kWarpThreads + laneIdx * kPackSize;
                    const int packOffset = packIdx * kPackSize;

                    if (gx < nx)
                    {
                        vecLdSt<kPackSize>(rowBuf + packOffset, dst + gy * nx + gx);
                    }
                }
            }
        }
    }
}


template <typename T>
struct Equal
{
    __host__ __device__
    inline bool operator()(const T & a, const T & b) = delete;
};


template <>
struct Equal<float>
{
    __host__ __device__
    inline bool operator()(float a, float b)
    {
        return abs(a - b) < kAbsTol + kRelTol * abs(b);
    }

    static constexpr float kAbsTol = 1e-4f;
    static constexpr float kRelTol = 1e-4f;
};


template <bool kDebugOutput = true>
void checkResult(const float * __restrict__ res,
                 const float * __restrict__ gt,
                 int nx,
                 int ny)
{
    static Equal<float> equal;

    bool correct = true;

    for (int i = 0; i < nx * ny; ++i)
    {
        if (!equal(res[i], gt[i]))
        {
            correct = false;
            break;
        }
    }

    printf("result is %s\n", correct ? "correct." : "WRONG!!!");

    if constexpr (kDebugOutput)
    {
        if (correct)
        {
            return;
        }

        printf("res:\n");

        for (int y = 0; y < 2; ++y)
        {
            for (int x = 0; x < nx; ++x)
            {
                printf("%11.6f ", res[y * nx + x]);
            }

            printf("\n");
        }

        printf("\n\ngt :\n");

        for (int y = 0; y < 2; ++y)
        {
            for (int x = 0; x < nx; ++x)
            {
                printf("%11.6f ", gt[y * nx + x]);
            }
            printf("\n");
        }

        printf("\n");
    }
}


int main(int argc, char * argv[])
{
    /// Switches for debugging output correctness.
    /// \param kDup        Set to 1 to debug output (kernel only launched once) and results will be checked.
    ///                    Set to values greater than 1 to profile.
    ///                    In the latter case, results will NOT be checked because it's in-place GEMM.
    ///                    We do not dispatch by build type because we have -G flag for Debug builds
    ///                    (that's for debugging runtime errors).
    /// \param kRandInput  Whether we random input.
    ///                    Enable when checking correctness or profiling.
    ///                    Disable when debugging output.
    constexpr int kDup = 1;
    constexpr bool kRandInput = true;

    constexpr bool kTestSoftmax = true;

    int nx = 1024;
    int ny = 1000;
    thrust::host_vector<float> hostSrc(ny * nx, 1.0f);
    thrust::host_vector<float> hostDst;

    if constexpr (kRandInput)
    {
        unsigned seed = std::random_device()();
        std::default_random_engine e(seed);
        std::normal_distribution<float> d(4.0f, 1.0f);
        // std::uniform_int_distribution<int> d(1, 10);
        auto g = [&e, &d]() -> float { return d(e); };
        std::generate(hostSrc.begin(), hostSrc.end(), g);
    }

    thrust::host_vector<float> gt(ny * nx, 1.0f);
    cpuSoftmax(hostSrc.data(), gt.data(), nx, ny);

    thrust::device_vector<float> devSrc = hostSrc;
    thrust::device_vector<float> devDst(ny * nx);

    // CUDA resources that require manual destruction.
    float ms;
    hipEvent_t ss, ee;
    CUDA_CHECK(hipEventCreate(&ss));
    CUDA_CHECK(hipEventCreate(&ee));

    constexpr int kPackSize = 4;
    constexpr int kWarpThreads = 32;

    constexpr dim3 kBlock(32, 8);
    constexpr int kThreadSpanX = 32;
    constexpr int kThreadSpanY = 1;
    constexpr int kBlockSpanX = kThreadSpanX * kBlock.x;
    constexpr int kBlockSpanY = kThreadSpanY * kBlock.y;

    dim3 grid((nx + kBlockSpanX - 1) / kBlockSpanX, (ny + kBlockSpanY - 1) / kBlockSpanY);

    // Test
    if constexpr (kTestSoftmax)
    {
        if constexpr (1 < kDup)
        {
            softmax<kBlock.x, kBlock.y, kThreadSpanX, kThreadSpanY, kPackSize, kWarpThreads><<<grid, kBlock>>>(
                    thrust::raw_pointer_cast(devSrc.data()),
                    thrust::raw_pointer_cast(devDst.data()),
                    nx,
                    ny
            );
            CUDA_CHECK(hipDeviceSynchronize());
        }

        CUDA_CHECK(hipEventRecord(ss));

        for (int dup = 0; dup < kDup; ++dup)
        {
            softmax<kBlock.x, kBlock.y, kThreadSpanX, kThreadSpanY, kPackSize, kWarpThreads><<<grid, kBlock>>>(
                    thrust::raw_pointer_cast(devSrc.data()),
                    thrust::raw_pointer_cast(devDst.data()),
                    nx,
                    ny
            );
        }

        CUDA_CHECK_LAST_ERROR();
        CUDA_CHECK(hipDeviceSynchronize());
        CUDA_CHECK(hipEventRecord(ee));
        CUDA_CHECK(hipEventSynchronize(ee));

        hostDst = devDst;

        std::printf("softmax: ");
        CUDA_CHECK(hipEventElapsedTime(&ms, ss, ee));
        std::printf("took %f ms, ", ms / kDup);

        if constexpr (1 == kDup)
        {
            checkResult(hostDst.data(), gt.data(), nx, ny);
        }
        else
        {
            std::printf("\n");
        }
    }

    // Free cuda resources.
    CUDA_CHECK(hipEventDestroy(ss));
    CUDA_CHECK(hipEventDestroy(ee));

    return EXIT_SUCCESS;
}

// Commonly-used Nsight Compute metrics:
// Bank conflicts:
//     l1tex__data_bank_conflicts_pipe_lsu_mem_shared_op_ld.sum
//     l1tex__data_bank_conflicts_pipe_lsu_mem_shared_op_st.sum
// Warp divergence:
//     smsp__thread_inst_executed_per_inst_executed
// L1/L2 cache hit rates:
//     l1tex__t_sector_hit_rate
//     lts__t_sector_hit_rate
// Global load/store throughputs and efficiencies:
//     l1tex__t_bytes_pipe_lsu_mem_global_op_ld.sum.per_second
//     l1tex__t_bytes_pipe_lsu_mem_global_op_st.sum.per_second
//     smsp__sass_average_data_bytes_per_sector_mem_global_op_ld.pct
//     smsp__sass_average_data_bytes_per_sector_mem_global_op_st.pct
// Achieved and therotical occupancy:
//     sm__warps_active.avg.pct_of_peak_sustained_active
//     sm__maximum_warps_per_active_cycle_pct

/*
# Profile bank conflicts:
ncu -k regex:gemmSmem --metrics l1tex__data_bank_conflicts_pipe_lsu_mem_shared_op_ld.sum,l1tex__data_bank_conflicts_pipe_lsu_mem_shared_op_st.sum ./cmake-build-release/demo
*/
