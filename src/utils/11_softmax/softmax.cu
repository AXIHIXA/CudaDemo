#include "hip/hip_runtime.h"
#include <algorithm>
#include <chrono>
#include <cmath>
#include <cstdlib>
#include <iostream>
#include <numeric>
#include <random>
#include <vector>

#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "utils/cuda_utils.h"


/// Softmax on innermost dimension.
void cpuSoftmax(const float * __restrict__ in, float * __restrict__ out, int nx, int ny)
{
    for (int y = 0; y < ny; ++y)
    {
        float rowSum = 0.0f;
        float rowMax = std::numeric_limits<float>::min();

        for (int x = 0; x < nx; ++x)
        {
            rowMax = std::max(in[y * nx + x], rowMax);
        }

        for (int x = 0; x < nx; ++x)
        {
            rowSum += std::exp(in[y * nx + x] - rowMax);
        }

        for (int x = 0; x < nx; ++x)
        {
            out[y * nx + x] = std::exp(in[y * nx + x] - rowMax) / rowSum;
        }
    }
}


template <typename T>
struct Max
{
    __device__ __forceinline__ constexpr bool operator()(const T & a, const T & b)
    {
        return (a < b) ? b : a;
    }
};


template <typename T>
struct Sum
{
    __device__ __forceinline__ T operator()(const T & a, const T & b)
    {
        return a + b;
    }
};


/// Bufferfly warp reduction.
/// See https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#reduction-across-a-warp
template <template <typename> class ReductionOp, typename T, int kWarpThreads = 32>
__inline__ __device__ T warpReduce(T val)
{
    #pragma unroll
    for (int mask = kWarpThreads >> 1; 0 < mask; mask >>= 1)
    {
        val = ReductionOp<T>()(val, __shfl_xor_sync(0xffffffff, val, mask, kWarpThreads));
    }

    return val;
}


// Used for vectorized stores and loads.
template <typename T, int kSize>
struct alignas(sizeof(T) * kSize) Vec
{
    T val[kSize];
};


// Vectorized load, from src's y-th row at column x, to dst.
template <int kPackSize>
__device__ void vecLoad(const float * src, int y, int nx, int x, float * dst)
{
    using Vec = Vec<float, kPackSize>;
    const int offset = (y * nx + x) / kPackSize;
    *reinterpret_cast<Vec *>(dst) = *(reinterpret_cast<const Vec *>(src) + offset);
}


// Vectorized store, from src, to dst's y-th row at column x.
template <int kPackSize>
__device__ void vecStore(const float * src, float * dst, int y, int nx, int x)
{
    using Vec = Vec<float, kPackSize>;
    const int offset = (y * nx + x) / kPackSize;
    *(reinterpret_cast<Vec *>(dst) + offset) = *reinterpret_cast<const Vec *>(src);
}


template <int kBlockDimX, int kBlockDimY, int kBlockSpanX, int kBlockSpanY, int kPackSize, int kWarpThreads = 32>
__global__ void softmax(const float * __restrict__ src,
                        float * __restrict__ dst,
                        int nx,
                        int ny)
{
    constexpr float kMinusInfinity = -10000000000.0f;

    constexpr int kThreadSpanX = kBlockSpanX / kBlockDimX;
    constexpr int kThreadSpanY = kBlockSpanY / kBlockDimY;
    constexpr int kNumPacks = kThreadSpanX / kPackSize;

    // Each warp handles a complete line of input.
    assert(nx <= kThreadSpanX * kWarpThreads);
    static_assert(kBlockDimX == kWarpThreads && kWarpThreads == 32);

    // Number of packs (vectorized load granularity on x dimension) should be integer.
    static_assert(kThreadSpanX % kPackSize == 0);

    const int tid = threadIdx.y * kBlockDimX + threadIdx.x;
    const int laneIdx = threadIdx.x;
    const int globalWarpIdx = blockIdx.y * blockDim.y + threadIdx.y;  // Each row constitutes a warp.
    const int globalWarps = gridDim.y * blockDim.y;  // Total number of warps in the whole grid.
    const int yStep = globalWarps * kThreadSpanY;

    float buf[kThreadSpanY][kThreadSpanX];

    // Warp view.
    for (int baseY = globalWarpIdx * kThreadSpanY; baseY < ny; baseY += yStep)
    {
        float threadMax[kThreadSpanY];

        // Warp view, each thread processes a row at a time.
        // Each thread reads part of a row into register and perform max reduciton.
        for (int rowIdx = 0; rowIdx < kThreadSpanY; ++rowIdx)
        {
            // Current row: baseY + rowIdx (same row for all threads in a warp).
            threadMax[rowIdx] = kMinusInfinity;
            float * rowBuf = buf[rowIdx];

            // Each threads loads kThreadSpanX elements into its rowBuf (GMEM -> REG),
            // loads are done by kNumPacks vectorized loads, each pack of length kPackSize.
            // Note that different packs of a thread are STRIDED
            // (to preserve a coalesced GMEM access footprint at warp scale).
            for (int packIdx = 0; packIdx < kNumPacks; ++packIdx)
            {
                const int packOffset = packIdx * kPackSize;
                const int x = (packIdx * kWarpThreads + laneIdx) * kPackSize;

                if (x < nx)
                {
                    vecLoad<kPackSize>(src, baseY + rowIdx, nx, x, rowBuf + packOffset);

                    for (int i = 0; i < kPackSize; ++i)
                    {
                        threadMax[rowIdx] = max(threadMax[rowIdx], rowBuf[packOffset + i]);
                    }
                }
                else
                {
                    for (int i = 0; i < kPackSize; ++i)
                    {
                        rowBuf[packOffset + i] = kMinusInfinity;
                    }
                }
            }
        }

        // Warp max aka row max.
        float rowMax[kThreadSpanY];

        for (int rowIdx = 0; rowIdx < kThreadSpanY; ++rowIdx)
        {
            rowMax[rowIdx] = warpReduce<Max, float>(threadMax[rowIdx]);
        }

        // Thread sum needs to be calculated after row max is reduced.
        // Modify in-place of registers from xi to exp(xi - xMax).
        float threadSum[kThreadSpanY];

        for (int rowIdx = 0; rowIdx < kThreadSpanY; ++rowIdx)
        {
            threadSum[rowIdx] = 0;
            float * rowBuf = buf[rowIdx];

            for (int i = 0; i < kThreadSpanX; ++i)
            {
                rowBuf[i] = exp(rowBuf[i] - rowMax[rowIdx]);
                threadSum[rowIdx] += rowBuf[i];
            }
        }

        float rowSum[kThreadSpanY];

        for (int rowIdx = 0; rowIdx < kThreadSpanY; ++rowIdx)
        {
            rowSum[rowIdx] = warpReduce<Sum, float>(threadSum[rowIdx]);
        }

        for (int rowIdx = 0; rowIdx < kThreadSpanY; ++rowIdx)
        {
            float * rowBuf = buf[rowIdx];

            for (int i = 0; i < kThreadSpanX; ++i)
            {
                rowBuf[i] = rowBuf[i] / rowSum[rowIdx];
            }

            for (int i = 0; i < kThreadSpanX; ++i)
            {
                const int x = (i * kWarpThreads + laneIdx) * kPackSize;

                if (x < nx)
                {
                    vecStore<kPackSize>(rowBuf + i * kPackSize, dst, baseY + rowIdx, nx, x);
                }
            }
        }
    }
}


template <typename T>
struct Equal
{
    __host__ __device__
    inline bool operator()(const T & a, const T & b) = delete;
};


template <>
struct Equal<float>
{
    __host__ __device__
    inline bool operator()(float a, float b)
    {
        return abs(a - b) < kEps;
    }

    static constexpr float kEps = 1e-3f;
};


template <bool kDebugOutput = true>
void checkResult(const float * __restrict__ res,
                 const float * __restrict__ gt,
                 int nx,
                 int ny)
{
    static Equal<float> equal;

    bool correct = true;

    for (int i = 0; i < nx * ny; ++i)
    {
        if (!equal(res[i], gt[i]))
        {
            correct = false;
            break;
        }
    }

    printf("result is %s\n", correct ? "correct." : "WRONG!!!");

    if constexpr (kDebugOutput)
    {
        printf("res:\n");

        for (int y = 0; y < 2; ++y)
        {
            for (int x = 0; x < nx; ++x)
            {
                printf("%11.6f ", res[y * nx + x]);
            }

            printf("\n");
        }

        printf("\n\ngt :\n");

        for (int y = 0; y < 2; ++y)
        {
            for (int x = 0; x < nx; ++x)
            {
                printf("%11.6f ", gt[y * nx + x]);
            }
            printf("\n");
        }

        printf("\n");
    }
}


int main(int argc, char * argv[])
{
    /// Switches for debugging output correctness.
    /// \param kDup        Set to 1 to debug output (kernel only launched once) and results will be checked.
    ///                    Set to values greater than 1 to profile.
    ///                    In the latter case, results will NOT be checked because it's in-place GEMM.
    ///                    We do not dispatch by build type because we have -G flag for Debug builds
    ///                    (that's for debugging runtime errors).
    /// \param kRandInput  Whether we random input.
    ///                    Enable when checking correctness or profiling.
    ///                    Disable when debugging output.
    constexpr int kDup = 1;
    constexpr bool kRandInput = true;

    constexpr bool kTestSoftmax = true;

    int nx = 1024;
    int ny = 1000;
    thrust::host_vector<float> hostSrc(ny * nx, 1.0f);
    thrust::host_vector<float> hostDst;

    if constexpr (kRandInput)
    {
        unsigned seed = std::random_device()();
        std::default_random_engine e(seed);
        std::normal_distribution<float> d(4.0f, 1.0f);
        // std::uniform_int_distribution<int> d(1, 10);
        auto g = [&e, &d]() -> float { return d(e); };
        std::generate(hostSrc.begin(), hostSrc.end(), g);
    }

    thrust::host_vector<float> gt(ny * nx, 1.0f);
    cpuSoftmax(hostSrc.data(), gt.data(), nx, ny);

    thrust::device_vector<float> devSrc = hostSrc;
    thrust::device_vector<float> devDst(ny * nx);

    // CUDA resources that require manual destruction.
    float ms;
    hipEvent_t ss, ee;
    CUDA_CHECK(hipEventCreate(&ss));
    CUDA_CHECK(hipEventCreate(&ee));

    constexpr int kPackSize = 1;
    constexpr int kWarpThreads = 32;
    constexpr dim3 kBlock(32, 8);
    constexpr int kBlockSpanX = kBlock.x * 32;
    constexpr int kBlockSpanY = kBlock.y;
    dim3 grid((nx + kBlockSpanX - 1) / kBlockSpanX, (ny + kBlockSpanY - 1) / kBlockSpanY);

    // Test
    if constexpr (kTestSoftmax)
    {
        if constexpr (1 < kDup)
        {
            softmax<kBlock.x, kBlock.y, kBlockSpanX, kBlockSpanY, kPackSize, kWarpThreads><<<grid, kBlock>>>(
                    thrust::raw_pointer_cast(devSrc.data()),
                    thrust::raw_pointer_cast(devDst.data()),
                    nx,
                    ny
            );
            CUDA_CHECK(hipDeviceSynchronize());
        }

        CUDA_CHECK(hipEventRecord(ss));

        for (int dup = 0; dup < kDup; ++dup)
        {
            softmax<kBlock.x, kBlock.y, kBlockSpanX, kBlockSpanY, kPackSize, kWarpThreads><<<grid, kBlock>>>(
                    thrust::raw_pointer_cast(devSrc.data()),
                    thrust::raw_pointer_cast(devDst.data()),
                    nx,
                    ny
            );
        }

        CUDA_CHECK_LAST_ERROR();
        CUDA_CHECK(hipDeviceSynchronize());
        CUDA_CHECK(hipEventRecord(ee));
        CUDA_CHECK(hipEventSynchronize(ee));

        hostDst = devDst;

        std::printf("softmax: ");
        CUDA_CHECK(hipEventElapsedTime(&ms, ss, ee));
        std::printf("took %f ms, ", ms / kDup);

        if constexpr (1 == kDup)
        {
            checkResult(hostDst.data(), gt.data(), nx, ny);
        }
        else
        {
            std::printf("\n");
        }
    }

    // Free cuda resources.
    CUDA_CHECK(hipEventDestroy(ss));
    CUDA_CHECK(hipEventDestroy(ee));

    return EXIT_SUCCESS;
}

// Commonly-used Nsight Compute metrics:
// Bank conflicts:
//     l1tex__data_bank_conflicts_pipe_lsu_mem_shared_op_ld.sum
//     l1tex__data_bank_conflicts_pipe_lsu_mem_shared_op_st.sum
// Warp divergence:
//     smsp__thread_inst_executed_per_inst_executed
// L1/L2 cache hit rates:
//     l1tex__t_sector_hit_rate
//     lts__t_sector_hit_rate
// Global load/store throughputs and efficiencies:
//     l1tex__t_bytes_pipe_lsu_mem_global_op_ld.sum.per_second
//     l1tex__t_bytes_pipe_lsu_mem_global_op_st.sum.per_second
//     smsp__sass_average_data_bytes_per_sector_mem_global_op_ld.pct
//     smsp__sass_average_data_bytes_per_sector_mem_global_op_st.pct
// Achieved and therotical occupancy:
//     sm__warps_active.avg.pct_of_peak_sustained_active
//     sm__maximum_warps_per_active_cycle_pct

/*
# Profile bank conflicts:
ncu -k regex:gemmSmem --metrics l1tex__data_bank_conflicts_pipe_lsu_mem_shared_op_ld.sum,l1tex__data_bank_conflicts_pipe_lsu_mem_shared_op_st.sum ./cmake-build-release/demo
*/
