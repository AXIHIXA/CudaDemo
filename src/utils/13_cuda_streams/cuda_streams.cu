#include "hip/hip_runtime.h"
#include <cassert>
#include <cmath>
#include <iostream>

#include <hip/hip_runtime.h>

#include "utils/cuda_utils.h"


template <typename T>
__global__ void add(T * x, T * y, T * z, int nx)
{
    int gx = blockDim.x * blockIdx.x + threadIdx.x;

    for (int i = gx; i < nx; i += gridDim.x * blockDim.x)
    {
        z[i] = y[i] + x[i];
    }
}

template <typename T>
void addCpu(T * x, T * y, T * z, int nx)
{
    for (int i = 0; i < nx; i++)
    {
        z[i] = y[i] + x[i];
    }
}


int main(int argc, char * argv[])
{
    using T = float;

    // Default stream took 1.933568 ms
    // Async stream took 1.511424 ms
    const int arraySize = 1000000;
    const int numBytes = arraySize * sizeof(T);
    const int numStreams = 4;
    const bool forceDefaultStream = true;

    assert(arraySize % numStreams == 0);
    const int nx = arraySize / numStreams;
    const int nxBytes = nx * sizeof(T);

    T * hx;
    T * hy;
    T * hz;

    // MUST manually alllocate pinned memory using hipHostAlloc
    CUDA_CHECK(hipHostAlloc(&hx, numBytes, hipHostMallocDefault));
    CUDA_CHECK(hipHostAlloc(&hy, numBytes, hipHostMallocDefault));
    CUDA_CHECK(hipHostAlloc(&hz, numBytes, hipHostMallocDefault));

    for (int i = 0; i < arraySize; ++i)
    {
        hx[i] = 1.0f;
        hy[i] = 1.0f;
    }

    auto hz_cpu = reinterpret_cast<T *>(malloc(numBytes));
    addCpu(hx, hy, hz_cpu, arraySize);

    T * dx;
    T * dy;
    T * dz;

    CUDA_CHECK(hipMalloc(&dx, numBytes));
    CUDA_CHECK(hipMalloc(&dy, numBytes));
    CUDA_CHECK(hipMalloc(&dz, numBytes));

    hipStream_t streams[numStreams];

    for (int i = 0; i < numStreams; ++i)
    {
        hipStreamCreate(&streams[i]);
    }

    const dim3 block(256);
    dim3 grid((nxBytes + block.x - 1) / block.x);

    float ms;
    hipEvent_t start;
    hipEvent_t stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    CUDA_CHECK(hipEventRecord(start));

    for (int i = 0; i < numStreams; ++i)
    {
        int offset = i * nx;

        hipMemcpyAsync(dx + offset,
                        hx + offset,
                        nxBytes,
                        hipMemcpyHostToDevice,
                        forceDefaultStream ? nullptr : streams[i]);

        hipMemcpyAsync(dy + offset,
                        hy + offset,
                        nxBytes,
                        hipMemcpyHostToDevice,
                        forceDefaultStream ? nullptr : streams[i]);

        add<<<grid, block, 0, forceDefaultStream ? nullptr : streams[i]>>>(
                dx + offset,
                dy + offset,
                dz + offset,
                nx);

        hipMemcpyAsync(hz + offset,
                        dz + offset,
                        nxBytes, hipMemcpyDeviceToHost,
                        forceDefaultStream ? nullptr : streams[i]);
    }

    // Use hipDeviceSynchronize to sync host and all streams of device.
    // When we only need sync one stream and device,
    // use hipStreamSynchronize, which is light-weight than multiple hipStreamSynchronize() s.
    hipDeviceSynchronize();

    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&ms, start, stop));
    printf("%s stream took %f ms\n", forceDefaultStream ? "Default" : "Async", ms);

    for (int i = 0; i < arraySize; ++i)
    {
        if (1e-6f < fabs(hz_cpu[i] - hz[i]))
        {
            printf("index: %d, cpu: %f, gpu: %f\n", i, hz_cpu[i], hz[i]);
            break;
        }
    }

    printf("Result right\n");

    for (int i = 0; i < numStreams; ++i)
    {
        hipStreamDestroy(streams[i]);
    }

    CUDA_CHECK(hipHostFree(hx));
    CUDA_CHECK(hipHostFree(hy));
    CUDA_CHECK(hipHostFree(hz));

    free(hz_cpu);

    CUDA_CHECK(hipFree(dx));
    CUDA_CHECK(hipFree(dy));
    CUDA_CHECK(hipFree(dz));

    return 0;
}